
#include <hip/hip_runtime.h>
__global__ void saxpy(float* Z, float A, float* X, float* Y) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  Z[id] = A * X[id] + Y[id];
}

