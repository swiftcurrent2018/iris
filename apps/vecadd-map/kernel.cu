
#include <hip/hip_runtime.h>
extern "C" __global__ void loop0(int* C, int* A, int* B) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  C[id] = A[id] + B[id];
}

extern "C" __global__ void loop1(int* D, int* C) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  D[id] = C[id] * 10;
}

extern "C" __global__ void loop2(int* E, int* D) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  E[id] = D[id] * 2;
}

