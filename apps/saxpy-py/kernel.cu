
#include <hip/hip_runtime.h>
extern "C" __global__ void saxpy0(float* Z, float A, float* X) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  Z[id] = A * X[id];
}

extern "C" __global__ void saxpy1(float* Z, float* Y) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  Z[id] += Y[id];
}

